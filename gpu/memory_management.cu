#include "gpu_common.h"
#include "memory_management.hh"

unsigned char * copyToGPUMemory(unsigned char * byte_arr, size_t byte_arr_size) {
    unsigned char * gpu_byte_arr;
    CHECK_CALL(hipMalloc(&gpu_byte_arr, byte_arr_size*sizeof(unsigned char)));
    CHECK_CALL(hipMemcpy(gpu_byte_arr, byte_arr, byte_arr_size*sizeof(unsigned char), hipMemcpyHostToDevice));
    return gpu_byte_arr;
}

unsigned int * copyToGPUMemory(unsigned int * byte_arr, size_t num_elements) {
    unsigned int * gpu_byte_arr;
    CHECK_CALL(hipMalloc(&gpu_byte_arr, num_elements*sizeof(unsigned int)));
    CHECK_CALL(hipMemcpy(gpu_byte_arr, byte_arr, num_elements*sizeof(unsigned int), hipMemcpyHostToDevice));
    return gpu_byte_arr;
}

void copyToGPUMemoryNoAlloc(unsigned int * gpuMem, unsigned int * input, size_t num_elements) {

    CHECK_CALL(hipMemcpy(gpuMem, input, num_elements*sizeof(unsigned int), hipMemcpyHostToDevice));
}

void copyToGPUMemoryNoAlloc(float * gpuMem, float * input, size_t num_elements) {

    CHECK_CALL(hipMemcpy(gpuMem, input, num_elements*sizeof(float), hipMemcpyHostToDevice));
}


void allocateGPUMem(size_t num_elements, unsigned int ** gpu_mem) {
    CHECK_CALL(hipMalloc(gpu_mem, num_elements*sizeof(unsigned int)));
}

void allocateGPUMem(size_t num_elements, float ** gpu_mem) {
    CHECK_CALL(hipMalloc(gpu_mem, num_elements*sizeof(float)));
}

void copyToHostMemory(float * gpu_mem, float * cpu_mem, size_t num_elements) {
    CHECK_CALL(hipMemcpy(cpu_mem, gpu_mem, num_elements*sizeof(float), hipMemcpyDeviceToHost));
}

void freeGPUMemory(unsigned char * gpu_ptr) {
    CHECK_CALL(hipFree(gpu_ptr));
}

void freeGPUMemory(float * gpu_ptr) {
    CHECK_CALL(hipFree(gpu_ptr));
}

void freeGPUMemory(unsigned int * gpu_ptr) {
    CHECK_CALL(hipFree(gpu_ptr));
}

void pinnedMemoryAllocator(unsigned int ** pinned_mem, size_t num_elements) {
    CHECK_CALL(hipHostAlloc((void **)pinned_mem, num_elements*sizeof(unsigned int), hipHostMallocDefault));
}
void pinnedMemoryAllocator(float ** pinned_mem, size_t num_elements) {
    CHECK_CALL(hipHostAlloc((void **)pinned_mem, num_elements*sizeof(float), hipHostMallocDefault));
}

void pinnedMemoryDeallocator(void * pinned_mem) {
    CHECK_CALL(hipHostFree(pinned_mem));
}
